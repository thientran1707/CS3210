/**
 * 
 * Matrix Multiplication - CUDA for GPUs
 *
 * CS3210
 *
 **/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <assert.h>

int size;

const int BLOCK_SIZE = 16;

typedef struct
{
	float ** element;
} matrix;


long long wall_clock_time()
{
#ifdef __linux__
	struct timespec tp;
	clock_gettime(CLOCK_REALTIME, &tp);
	return (long long)(tp.tv_nsec + (long long)tp.tv_sec * 1000000000ll);
#else
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (long long)(tv.tv_usec * 1000 + (long long)tv.tv_sec * 1000000000ll);
#endif
}

/**
 * Allocates memory for a matrix of size SIZE
 * The memory is allocated row-major order, i.e. 
 *  elements from the same row are allocated at contiguous 
 *  memory addresses.
 **/
void allocate_matrix(matrix* m)
{
	int i;
	hipError_t rc;
	
	// allocate array for all the rows
	rc = hipMallocManaged((void**)&(m->element), sizeof(float*) * size);
	if (rc != hipSuccess)
	{
		fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(rc));
		exit(1);
	}
	
	// allocate an array for each row of the matrix
	for (i = 0; i < size; i++)
	{
		rc = hipMallocManaged((void**)&(m->element[i]), sizeof(float) * size);
		if (rc != hipSuccess)
		{
			fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(rc));
			exit(1);
		}
	}
}

/**
 * Free the memory allocated for a matrix.
 **/
void free_matrix(matrix* m) {
	int i;
	for (i = 0; i < size; i++)
		hipFree(m->element[i]);
	hipFree(m->element);
}

/**
 * Initializes the elements of the matrix with
 * random values between 0 and 9
 **/
void init_matrix(matrix m)
{
	int i, j;
	
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
		{
			m.element[i][j] = rand() % 10;
		}
}

/**
 * Initializes the elements of the matrix with
 * element 0.
 **/
void init_matrix_zero(matrix m)
{
	int i, j;
	
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
		{
			m.element[i][j] = 0.0;
		}
}


/**
 * Multiplies matrix @a with matrix @b storing
 * the result in matrix @result
 * 
 * The multiplication algorithm is the O(n^3) 
 * algorithm
 */
void mm(matrix a, matrix b, matrix result)
{
	int i, j, k;
	
	// Do the multiplication
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
			for(k = 0; k < size; k++)
				result.element[i][j] += a.element[i][k] * b.element[k][j];
}

/**
 * Each kernel computes the result element (i,j).
 */
__global__ void mm_kernel(matrix a, matrix b, matrix result, int size)
{
        // Block index
        int bx = blockIdx.x;
        int by = blockIdx.y;

        // Thread index
        int tx = threadIdx.x;
        int ty = threadIdx.y;

        // begin of the first sub-matrix of a processed by the block
        int aBegin = size * BLOCK_SIZE * by;
        int aEnd = aBegin + size - 1;
        int aStep = BLOCK_SIZE;

        int bBegin = BLOCK_SIZE * bx;
        int bStep = BLOCK_SIZE * size;
        
        float Csub = 0;

        for (int _a = aBegin, _b = bBegin;
             _a <= aEnd;
             _a += aStep, _b += bStep) {
            
            __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
            __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
            int a_index = _a + size * ty + tx;
            int a_i = a_index / size;
            int a_j = a_index % size;

            int b_index = _b + size * ty + tx;
            int b_i = b_index / size;
            int b_j = b_index % size;

            As[ty][tx] = a_i < size ? a.element[a_i][a_j] : 0;
            Bs[ty][tx] = b_i < size ? b.element[b_i][b_j] : 0;

            __syncthreads();

            for (int k = 0; k < BLOCK_SIZE; k++) {
                Csub += As[ty][k] * Bs[k][tx];
            }

            __syncthreads();

        }

        int c_index = size * BLOCK_SIZE * by + BLOCK_SIZE * bx + size * ty + tx;
        int c_i = c_index / size;
        int c_j = c_index % size;
        result.element[c_i][c_j] = Csub;
}

void print_matrix(matrix m)
{
	int i, j;
	
	for (i = 0; i < size; i++)
	{
		printf("row %4d: ", i);
		for (j = 0; j < size; j++)
			printf("%6.2f  ", m.element[i][j]);
		printf("\n");
	}
}



void work()
{
	matrix a, b, result1, result2;

	long long before, after;
	int correct, i, j, dim;
	hipError_t rc;

	// Allocate memory for matrices
	allocate_matrix(&a);
	allocate_matrix(&b);
	allocate_matrix(&result1);
	allocate_matrix(&result2);

	// Initialize matrix elements
	init_matrix(a);
	init_matrix(b);

	before = wall_clock_time();
	mm(a, b, result1);
	after = wall_clock_time();
        fprintf(stderr, "Matrix multiplication on CPU took %1.2f seconds\n", ((float)(after - before))/1000000000);

	// Perform CUDA matrix  multiplication
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);			// a block of 32 x 32 CUDA threads
	dim = (size % BLOCK_SIZE == 0) ? size / BLOCK_SIZE : size / BLOCK_SIZE + 1; 
	dim3 grid(dim, dim);	// a grid of CUDA thread blocks

	before = wall_clock_time();
	mm_kernel<<<grid, block>>>(a, b, result2, size);
	hipDeviceSynchronize();
	after = wall_clock_time();

	fprintf(stderr, "Matrix multiplication on GPU took %1.2f seconds\n", ((float)(after - before))/1000000000);

	// was there any error?
        rc = hipGetLastError();
        if (rc != hipSuccess)
                printf("Last CUDA error %s\n", hipGetErrorString(rc));

	// Compare the results
	correct = 1;
	for (i = 0; correct && i < size; i++)
		for (j = 0; j < size; j++)
			if (result1.element[i][j] != result2.element[i][j]) {
				correct = 0;
				break;
			}

	if (correct)
		printf("The result matrices are identical!\n");
	else
		printf("Difference in result matrices at element (%d, %d)!\n", i, j);

	free_matrix(&a);
	free_matrix(&b);
	free_matrix(&result1);
	free_matrix(&result2);
}


int main(int argc, char ** argv)
{
	srand(0); 

	printf("Usage: %s <size>\n", argv[0]);
    
	if (argc >= 2)
		size = atoi(argv[1]);
	else
		size = 1024;
		
	fprintf(stderr,"Sequential matrix multiplication of size %d\n", size);
    
	// Multiply the matrices
	work();

	return 0;
}
